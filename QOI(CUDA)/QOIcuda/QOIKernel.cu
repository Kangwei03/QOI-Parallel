#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./QOIKernel.cuh"
#include <>
#include "qoi.h"
#include <stdio.h>
#include <malloc.h>
#include <string.h>
#include <algorithm>
#include <hip/hip_cooperative_groups.h>

#ifndef QOI_ZEROARR
#define QOI_ZEROARR(a) memset((a),0,sizeof(a))
#endif

static void qoi_write_32(unsigned char* bytes, int* p, unsigned int v) {
    bytes[(*p)++] = (0xff000000 & v) >> 24;
    bytes[(*p)++] = (0x00ff0000 & v) >> 16;
    bytes[(*p)++] = (0x0000ff00 & v) >> 8;
    bytes[(*p)++] = (0x000000ff & v);
}

#define QOI_OP_RUN   0xC0
#define QOI_MAGIC (((unsigned int)'q') << 24 | ((unsigned int)'o') << 16 | ((unsigned int)'i') << 8 | ((unsigned int)'f'))
#define QOI_OP_INDEX 0x00
#define QOI_OP_DIFF  0x40
#define QOI_OP_LUMA  0x80
#define QOI_OP_RGB   0xFE
#define QOI_OP_RGBA  0xFF
#define QOI_MASK_2   0xC0
#define QOI_COLOR_HASH(C) (((C.rgba.r * 3) + (C.rgba.g * 5) + (C.rgba.b * 7) + (C.rgba.a * 11)) % 64)

#ifndef QOI_HEADER_SIZE
#define QOI_HEADER_SIZE 14
#endif

#ifndef __builtin_bswap32
#define __builtin_bswap32(x) \
    ((((x) & 0xff000000u) >> 24) | \
     (((x) & 0x00ff0000u) >> 8) | \
     (((x) & 0x0000ff00u) << 8) | \
     (((x) & 0x000000ffu) << 24))
#endif

typedef union {
    struct { unsigned char r, g, b, a; } rgba;
    unsigned int v;
} qoi_rgba_t;

__device__ void atomicWriteUChar(unsigned char* addr, unsigned char value) {
    unsigned int* base_addr = (unsigned int*)(((size_t)addr) & ~3);
    unsigned int shift = ((size_t)addr & 3) * 8;
    unsigned int mask = 0xFF << shift;
    unsigned int old, assumed;

    old = *base_addr;
    do {
        assumed = old;
        old = atomicCAS(base_addr,
            assumed,
            (assumed & ~mask) | (value << shift));
    } while (assumed != old);
}

__global__ void qoi_encode_kernel(
    const unsigned char* data,
    unsigned char* output,
    int width,
    int height,
    int channels,
    int* total_size,
    unsigned char* shared_local_bytes,
    int* shared_num_bytes,
    int* total_chunks_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = width * height;
    int p = 0;
    int num_process = 512;

    if (idx >= total_pixels) return;

    qoi_rgba_t index[64];

    QOI_ZEROARR(index);

    int pixels_per_thread = (total_pixels + num_process - 1) / num_process;
    int start_pixel = idx * pixels_per_thread;
    int end_pixel = min(start_pixel + pixels_per_thread, total_pixels);
    int start = start_pixel * channels;
    int end = min(end_pixel * channels, total_pixels * channels);
    int chunk_size = end - start;

    unsigned char* local_bytes;

    qoi_rgba_t px, px_prev;

    if (idx < num_process) {
        local_bytes = &shared_local_bytes[idx * chunk_size];

        if (chunk_size > 0) {
            if (idx == 0)
            {
                px_prev.rgba.r = 0;
                px_prev.rgba.g = 0;
                px_prev.rgba.b = 0;
                px_prev.rgba.a = 255;
            }
            else if (channels == 3) {
                px_prev.rgba.r = data[start - 3];
                px_prev.rgba.g = data[start - 2];
                px_prev.rgba.b = data[start - 1];
                px_prev.rgba.a = 255;
            }
            else {
                px_prev.rgba.r = data[start - 4];
                px_prev.rgba.g = data[start - 3];
                px_prev.rgba.b = data[start - 2];
                px_prev.rgba.a = data[start - 1];
            }
            px = px_prev;



            int run = 0;

            for (int px_pos = start; px_pos < end && px_pos + channels <= total_pixels * channels; px_pos += channels) {
                if (p + 5 >= chunk_size) break;  // Ensure space for largest possible encoding

                px.rgba.r = data[px_pos + 0];
                px.rgba.g = data[px_pos + 1];
                px.rgba.b = data[px_pos + 2];
                px.rgba.a = (channels == 4) ? data[px_pos + 3] : 255;

                if (px.v == px_prev.v) {
                    run++;
                    if (run == 62 || px_pos == end - channels) {
                        local_bytes[p++] = QOI_OP_RUN | (run - 1);
                        run = 0;
                    }
                }
                else {
                    if (run > 0) {
                        local_bytes[p++] = QOI_OP_RUN | (run - 1);
                        run = 0;
                    }

                    if (px.rgba.a == px_prev.rgba.a) {
                        signed char vr = px.rgba.r - px_prev.rgba.r;
                        signed char vg = px.rgba.g - px_prev.rgba.g;
                        signed char vb = px.rgba.b - px_prev.rgba.b;
                        signed char vg_r = vr - vg;
                        signed char vg_b = vb - vg;

                        if (vr > -3 && vr < 2 && vg > -3 && vg < 2 && vb > -3 && vb < 2) {
                            local_bytes[p++] = QOI_OP_DIFF | ((vr + 2) << 4) | ((vg + 2) << 2) | (vb + 2);
                        }
                        else if (vg_r > -9 && vg_r < 8 && vg > -33 && vg < 32 && vg_b > -9 && vg_b < 8) {
                            local_bytes[p++] = QOI_OP_LUMA | (vg + 32);
                            local_bytes[p++] = (vg_r + 8) << 4 | (vg_b + 8);
                        }
                        else {
                            local_bytes[p++] = QOI_OP_RGB;
                            local_bytes[p++] = px.rgba.r;
                            local_bytes[p++] = px.rgba.g;
                            local_bytes[p++] = px.rgba.b;
                        }
                    }
                    else {
                        local_bytes[p++] = QOI_OP_RGBA;
                        local_bytes[p++] = px.rgba.r;
                        local_bytes[p++] = px.rgba.g;
                        local_bytes[p++] = px.rgba.b;
                        local_bytes[p++] = px.rgba.a;
                    }
                }
                px_prev = px;
            }

            // Ensure run is terminated at chunk end
            if (run > 0) {
                local_bytes[p++] = QOI_OP_RUN | (run - 1);
            }
        }

        shared_num_bytes[idx] = p;
        atomicAdd(total_size, p);
    }


    __syncthreads();

    if (idx < num_process)
    {

        int start_write = 0;

        for (int i = 0; i < idx; i++)
        {
            start_write += shared_num_bytes[i];
        }

        // Write chunk to global memory using a single atomic operation
        for (int i = 0; i < p; i++) {
            atomicWriteUChar(&output[start_write + i], local_bytes[i]);
        }



    }


}


void* qoi_encode_cuda(const void* data, const qoi_desc* desc, int* out_len) {
    if (!data || !desc || !out_len) return NULL;

    int total_pixels = desc->width * desc->height;
    int data_size = total_pixels * desc->channels;

    // Device memory
    unsigned char* d_data, * d_output, * d_shared_local_bytes;
    int* d_chunk_positions, * d_total_chunks, * d_shared_num_bytes, * d_total_chunks_size;

    // Allocate device memory
    hipMalloc(&d_data, data_size);
    hipMalloc(&d_output, total_pixels * 5); // Max 5 bytes per pixel
    hipMalloc(&d_chunk_positions, total_pixels * sizeof(int));
    hipMalloc(&d_total_chunks, sizeof(int));
    hipMalloc(&d_shared_local_bytes, sizeof(unsigned char) * data_size * 2);
    hipMalloc(&d_shared_num_bytes, sizeof(int) * 512);
    hipMalloc(&d_total_chunks_size, sizeof(int));


    // Copy input data to device
    hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);
    hipMemset(d_total_chunks, 0, sizeof(int));

    // Launch kernel
    int threadsPerBlock = 512;  // Multiple of warp size (32)
    int blocksPerGrid = (512 + threadsPerBlock - 1) / threadsPerBlock;

    qoi_encode_kernel << <blocksPerGrid, threadsPerBlock >> > (
        d_data,
        d_output,
        desc->width,
        desc->height,
        desc->channels,
        d_total_chunks,
        d_shared_local_bytes,
        d_shared_num_bytes,
        d_total_chunks_size
        );

    // Get total chunks
    int total_chunks;
    hipMemcpy(&total_chunks, d_total_chunks, sizeof(int), hipMemcpyDeviceToHost);

    // Allocate and create final output with header
    int header_size = QOI_HEADER_SIZE;
    *out_len = header_size + total_chunks + 8; // +8 for end marker
    //printf("total chunkssssss = %d", total_chunks);
    unsigned char* output = (unsigned char*)malloc(*out_len);

    // Write header
    int p = 0;
    qoi_write_32(output, &p, QOI_MAGIC);
    qoi_write_32(output, &p, desc->width);
    qoi_write_32(output, &p, desc->height);
    output[p++] = desc->channels;
    output[p++] = desc->colorspace;

    // Copy encoded data
    hipMemcpy(output + p, d_output, total_chunks, hipMemcpyDeviceToHost);
    p += total_chunks;

    // Add end marker
    memcpy(output + p, "\0\0\0\0\0\0\0\1", 8);

    // Cleanup
    hipFree(d_data);
    hipFree(d_output);
    hipFree(d_chunk_positions);
    hipFree(d_total_chunks);
    hipFree(d_shared_local_bytes);
    hipFree(d_shared_num_bytes);
    hipFree(d_total_chunks_size);

    return output;
}

__global__ void decode_chunks_kernel(
    const unsigned char* encoded_data,
    unsigned char* decoded_data,
    int channels,
    int* px_len,
    int* d_p,
    int* size,
    unsigned char* shared_partial_pixel,
    int* shared_num_bytes
) {
    int px_pos;
    int run = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int p = 0;
    int num_threads = 16;
    qoi_rgba_t px = { 0, 0, 0, 255 };  // Initialize default values 

    int chunks_len = *size - 8;

    int pixels_per_thread = (*px_len + num_threads - 1) / num_threads;
    unsigned char* partial_pixel = &shared_partial_pixel[idx * pixels_per_thread * channels];

    int chunk_size = *size / num_threads;
    int start = chunk_size * idx;


        px = { 0, 0, 0, 255 };
    

    int end = start + chunk_size;
    if (end < chunks_len && idx == num_threads - 1)
    {
        end = chunks_len;
    }

    if (idx < num_threads)
    {

        if (idx == 0) {
            p = 0;  // Start from header offset for first thread
        }
        else {
            // Calculate approximate start position based on previous chunk
            int encoded_chunk_size = *size / num_threads;
            p = encoded_chunk_size * idx;

            // Back up to find a valid opcode
            while (p > 0 &&
                encoded_data[p] != QOI_OP_RGB &&
                encoded_data[p] != QOI_OP_RGBA) {
                p--;
            }
        }

        while (encoded_data[end] != QOI_OP_RGB && encoded_data[end] != QOI_OP_RGBA && idx != num_threads - 1)
        {
            end--;

        };



        int partialPixelsSize = 0;

        for (px_pos = 0; px_pos < *px_len; px_pos += channels)
        {

            if (run > 0) {
                run--;
            }
            else if (p < end) {
                int b1 = encoded_data[p++];

                if (b1 == QOI_OP_RGB) {
                    px.rgba.r = encoded_data[p++];
                    px.rgba.g = encoded_data[p++];
                    px.rgba.b = encoded_data[p++];
                }
                else if (b1 == QOI_OP_RGBA) {
                    px.rgba.r = encoded_data[p++];
                    px.rgba.g = encoded_data[p++];
                    px.rgba.b = encoded_data[p++];
                    px.rgba.a = encoded_data[p++];
                }
                else if ((b1 & QOI_MASK_2) == QOI_OP_DIFF) {

                    px.rgba.r += ((b1 >> 4) & 0x03) - 2;
                    px.rgba.g += ((b1 >> 2) & 0x03) - 2;
                    px.rgba.b += (b1 & 0x03) - 2;
                }
                else if ((b1 & QOI_MASK_2) == QOI_OP_LUMA) {

                    int b2 = encoded_data[p++];
                    int vg = (b1 & 0x3f) - 32;
                    px.rgba.r += vg - 8 + ((b2 >> 4) & 0x0f);
                    px.rgba.g += vg;
                    px.rgba.b += vg - 8 + (b2 & 0x0f);
                }
                else if ((b1 & QOI_MASK_2) == QOI_OP_RUN) {
                    run = (b1 & 0x3f);
                }

            }

            partial_pixel[px_pos + 0] = px.rgba.r;
            partial_pixel[px_pos + 1] = px.rgba.g;
            partial_pixel[px_pos + 2] = px.rgba.b;
            partialPixelsSize += 3;

            if (channels == 4) {
                partial_pixel[px_pos + 3] = px.rgba.a;
                partialPixelsSize++;
            }

            if (p >= end && run == 0)
            {
                break;
            }
        }
        shared_num_bytes[idx] = partialPixelsSize;
    }

    __syncthreads();


    if (idx < num_threads)
    {

        int start_write = 0;

        for (int i = 0; i < idx; i++)
        {
            start_write += shared_num_bytes[i];
        }

        // Write chunk to global memory using a single atomic operation
        for (int i = 0; i < shared_num_bytes[idx]; i++) {
            atomicWriteUChar(&decoded_data[start_write + i], partial_pixel[i]);
        }



    }
}

void* qoi_decode_cuda(const void* data, int size, qoi_desc* desc, int channels) {
    if (!data || !desc || size < QOI_HEADER_SIZE + 8) {
        return NULL;
    }

    const unsigned char* bytes = (const unsigned char*)data;
    int p = 0;

    // Read header
    unsigned int magic = (bytes[p] << 24) | (bytes[p + 1] << 16) | (bytes[p + 2] << 8) | bytes[p + 3];
    p += 4;
    desc->width = (bytes[p] << 24) | (bytes[p + 1] << 16) | (bytes[p + 2] << 8) | bytes[p + 3];
    p += 4;
    desc->height = (bytes[p] << 24) | (bytes[p + 1] << 16) | (bytes[p + 2] << 8) | bytes[p + 3];
    p += 4;
    desc->channels = bytes[p++];
    desc->colorspace = bytes[p++];

    if (channels == 0) {
        channels = desc->channels;
    }

    // Validate header
    if (magic != QOI_MAGIC || desc->width == 0 || desc->height == 0 ||
        desc->channels < 3 || desc->channels > 4 || desc->colorspace > 1) {
        return NULL;
    }

    int px_len = desc->width * desc->height * channels;
    int encoded_size = size - QOI_HEADER_SIZE - 8;  // Subtract header and end marker

    // Allocate device memory
    unsigned char* d_encoded, * d_decoded, * d_partial_pixels;
    int* d_ps_len, * d_p, * d_size, * d_shared_num_bytes;

    hipMalloc(&d_encoded, encoded_size);
    hipMalloc(&d_decoded, px_len);
    hipMalloc(&d_ps_len, sizeof(int));
    hipMalloc(&d_p, sizeof(int));
    hipMalloc(&d_size, sizeof(int));
    hipMalloc(&d_partial_pixels, px_len * 5);
    hipMalloc(&d_shared_num_bytes, sizeof(int) * 16);


    // Copy encoded data to device
    hipMemcpy(d_encoded, bytes + QOI_HEADER_SIZE, encoded_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ps_len, &px_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_p, &p, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernels
    int threadsPerBlock = 16;  // Match your num_threads
    int blocksPerGrid = (16 + threadsPerBlock - 1) / threadsPerBlock;

    decode_chunks_kernel << <blocksPerGrid, threadsPerBlock >> > (
        d_encoded,
        d_decoded,
        channels,
        d_ps_len,
        d_p,
        d_size,
        d_partial_pixels,
        d_shared_num_bytes
        );

    // Allocate and copy result
    unsigned char* result = (unsigned char*)malloc(px_len);
    hipMemcpy(result, d_decoded, px_len, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_encoded);
    hipFree(d_decoded);
    hipFree(d_ps_len);
    hipFree(d_p);
    hipFree(d_size);
    hipFree(d_partial_pixels);
    hipFree(d_shared_num_bytes);

    return result;
}